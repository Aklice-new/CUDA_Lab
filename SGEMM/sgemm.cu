#include <__clang_cuda_builtin_vars.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#include "assert.h"

#define FETCH_FLOAT4(pointer) (reinterpret_cast<float4*>(&pointer)[0])
#define OFFSET(row, col, ld) ((row * ld) + col)
#define CHECK_CUDA_ERROR(func)                                                 \
  {                                                                            \
    hipError_t e = (func);                                                    \
    if (e != hipSuccess) {                                                    \
      printf("%s %d CUDA : %s \n", __FILE__, __LINE__, hipGetErrorString(e)); \
    }                                                                          \
  }
template <const int BLOCK_SIZE_M,           // block size M bm = 128
          const int BLOCK_SIZE_N,           // block size N bn = 128
          const int BLOCK_SIZE_K,           // block size K bk = 8
          const int THREAD_SIZE_Y,          // thread size rm = 8
          const int THREAD_SIZE_X,          // thread size rn = 8
          const bool ENABLE_DOUBLE_BUFFER>  // enable double buffer ?
__global__ void sgemm(float* __restrict__ A, float* __restrict__ B,
                      float* __restrict__ C, const int M, const int N,
                      const int K) {
  // block块的坐标
  int bx = blockIdx.x;
  int by = blockIdx.y;
  // thread的坐标
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  // 每个block中线程横向和纵向的线程个数
  const int THREAD_X_PER_BLOCK = BLOCK_SIZE_N / THREAD_SIZE_X;
  const int THREAD_Y_PER_BLOCK = BLOCK_SIZE_M / THREAD_SIZE_Y;
  const int THREAD_NUM_PER_BLOCK = THREAD_X_PER_BLOCK * THREAD_Y_PER_BLOCK;

  int tid = ty * THREAD_X_PER_BLOCK + tx;

  // 为计算改block所申请的共享内存的空间，2倍大小是为了做double buffer
  __shared__ float A_s[2][BLOCK_SIZE_K][BLOCK_SIZE_M];
  __shared__ float B_s[2][BLOCK_SIZE_K][BLOCK_SIZE_N];
  // 该线程负责计算的一小块 rm * rn 寄存器类型
  float accum[THREAD_SIZE_Y][THREAD_SIZE_X] = {0};

  // frag_a rm * 1  frag_b  1 * rn 寄存器类型
  float frag_a[2][THREAD_SIZE_Y];
  float frag_b[2][THREAD_SIZE_X];

  // 首先要从global memory中把数据加载到 shared memory中
  // shared memory size = (bm * bk) = 128 * 8
  // 一个block中256个线程，则每个线程负责 128 * 8 / 256= 4 个元素的加载
  // A_s,B_s来说每一行需要多少个线程来加载元素，这里采用的是float4类型对数据进行读取
  // 即一次性读入4个元素
  const int A_TILE_THREAD_PER_ROW = BLOCK_SIZE_K / 4;
  const int B_TILE_THREAD_PER_ROW = BLOCK_SIZE_N / 4;

  // 计算该线程负责读取的tile的row的位置
  const int A_TILE_ROW_START = tid / A_TILE_THREAD_PER_ROW;
  const int B_TILE_ROW_START = tid / B_TILE_THREAD_PER_ROW;
  // 然后计算该线程读取的tile的列的位置, tid表示的是block的线程索引，分别得到A，
  // B 中的列的索引
  const int A_TILE_COL = tid % A_TILE_THREAD_PER_ROW * 4;
  const int B_TILE_COL = tid % B_TILE_THREAD_PER_ROW * 4;
  // block中线程的总数除以读取一行所需要的线程数得到步长，因为可能一次性读不完所有数据
  const int A_TILE_ROW_STRIDE = THREAD_NUM_PER_BLOCK / A_TILE_THREAD_PER_ROW;
  const int B_TILE_ROW_STRIDE = THREAD_NUM_PER_BLOCK / B_TILE_THREAD_PER_ROW;

  // 通过block的位置来定位到A B矩阵的对应行和对应列
  A = &A[(BLOCK_SIZE_M * by) * K];
  B = &B[BLOCK_SIZE_N * bx];
  // 首先将数据加载到A_s和B_s中来， 这时第一次加载，全部都加载到0中
  // double buffer
#pragma unroll
  for (int i = 0; i < BLOCK_SIZE_M; i += A_TILE_ROW_STRIDE) {
    // int ldg_index = i / A_TILE_ROW_STRIDE * 4;
    // 下面这部分是将本来 bm * bk 的区域进行转置，读入到 bk * bm 的区域中来
    float4 data = FETCH_FLOAT4(A[OFFSET(A_TILE_ROW_START + i, A_TILE_COL, K)]);
    A_s[0][A_TILE_COL][A_TILE_ROW_START + i] = data.x;
    A_s[0][A_TILE_COL + 1][A_TILE_ROW_START + i] = data.y;
    A_s[0][A_TILE_COL + 2][A_TILE_ROW_START + i] = data.z;
    A_s[0][A_TILE_COL + 3][A_TILE_ROW_START + i] = data.w;
  }
#pragma unroll
  for (int i = 0; i < BLOCK_SIZE_N; i += B_TILE_ROW_STRIDE) {
    // 读入B矩阵的时候不需要进行转置
    FETCH_FLOAT4(B_s[0][B_TILE_ROW_START + i][B_TILE_COL]) =
        FETCH_FLOAT4(B[OFFSET(B_TILE_ROW_START + i, B_TILE_COL, N)]);
  }
  __syncthreads();  // 保证数据加载完成

// 将该线程负责计算的 rm * rn区域内所需要的数据从A_s中加载到寄存器frag_a
#pragma unroll
  // 通过float4类型进行读取的，所以每次加4， THREAD_SIZE_Y 就是 rm
  for (int thread_y = 0; thread_y < THREAD_SIZE_Y; thread_y += 4) {
    // 这里因为A_s是转置存储的，所以本来需要读取A_s的一列，现在变为读取一行了
    // 这里是从第一列（行）开始读取的所以A_s第一维是0
    // THREAD_SIZE_Y * ty 表示的是当前这个线程负责的那一片
    // rm * rn 的区域的起始位置
    FETCH_FLOAT4(frag_a[0][thread_y]) =
        FETCH_FLOAT4(A_s[0][0][THREAD_SIZE_Y * ty + thread_y]);
  }
#pragma unroll
  for (int thread_x = 0; thread_x < THREAD_SIZE_X; thread_x += 4) {
    FETCH_FLOAT4(frag_b[0][thread_x]) =
        FETCH_FLOAT4(B_s[0][0][THREAD_SIZE_X * tx + thread_x]);
  }

  int write_stage_idx = 1;
  int tile_idx = 0;
  do {
    tile_idx += BLOCK_SIZE_K;
    // double buffer 从全局内存中加载下一个tile  大循环
    // 大循环是指，为了计算一个block bm * bn
    // 将A矩阵的 bm * K 和B矩阵的 K * bn分别划分成 bm * bk 和 bk * bn 个小块
    // 大循环里就是负责计算这些小块里的数据
    // double buffer
    // load next A_s
    if (tile_idx < K) {
#pragma unroll
      for (int i = 0; i < BLOCK_SIZE_M; i += A_TILE_ROW_STRIDE) {
        float4 data = FETCH_FLOAT4(
            A[OFFSET(A_TILE_ROW_START + i, A_TILE_COL + tile_idx, K)]);
        A_s[write_stage_idx][A_TILE_COL][A_TILE_ROW_START + i] = data.x;
        A_s[write_stage_idx][A_TILE_COL + 1][A_TILE_ROW_START + i] = data.y;
        A_s[write_stage_idx][A_TILE_COL + 2][A_TILE_ROW_START + i] = data.z;
        A_s[write_stage_idx][A_TILE_COL + 3][A_TILE_ROW_START + i] = data.w;
      }
      // load next B_s
#pragma unroll
      for (int i = 0; i < BLOCK_SIZE_N; i += B_TILE_ROW_STRIDE) {
        FETCH_FLOAT4(B_s[write_stage_idx][B_TILE_ROW_START + i][B_TILE_COL]) =
            FETCH_FLOAT4(
                B[OFFSET(B_TILE_ROW_START + tile_idx + i, B_TILE_COL, N)]);
      }
    }

    int load_stage_idx = write_stage_idx ^ 1;

    // 下面的内容就是小循环内完成的工作，
    // 具体工作是完成从A_s 和 B_s中依次读入rm, rn个数据进入寄存器
    // 然后把A_s 和 B_s中的数据加载到寄存器中
    // 依次计算每一次 rm + rn 个数据对该线程负责的 rm * rn 个数据的贡献
#pragma unroll
    for (int j = 0; j < BLOCK_SIZE_K - 1; j++) {
      // load next tile from shared mem to register
      // from A_s to frag_a
      for (int thread_y = 0; thread_y < THREAD_SIZE_Y; thread_y += 4) {
        FETCH_FLOAT4(frag_a[(j + 1) % 2][thread_y]) = FETCH_FLOAT4(
            A_s[load_stage_idx][j + 1][THREAD_SIZE_Y * ty + thread_y]);
      }
      // from B_s to frag_b
      for (int thread_x = 0; thread_x < THREAD_SIZE_X; thread_x += 4) {
        FETCH_FLOAT4(frag_b[(j + 1) % 2][thread_x]) = FETCH_FLOAT4(
            B_s[load_stage_idx][j + 1][THREAD_SIZE_X * tx + thread_x]);
      }
// compute THREAD_SIZE_X * THREAD_SIZE_Y
#pragma unroll
      for (int thread_y = 0; thread_y < THREAD_SIZE_Y; thread_y++) {
#pragma unroll
        for (int thread_x = 0; thread_x < THREAD_SIZE_X; thread_x++) {
          // accum 在每一次的小循环中保存的只是完整解的一部分
          accum[thread_y][thread_x] +=
              frag_a[j % 2][thread_y] * frag_b[j % 2][thread_x];
        }
      }
    }

    // 为了保证所有线程都完成对它负责的区域的数据的计算 需进行同步
    __syncthreads();
    // double buffer
    write_stage_idx ^= 1;
    // 最后在完成最后一次小的迭代，至于为什么要将8次小迭代分为7次加1次，原文作者说这样是为了隐藏延迟
    // from A_s to frag_a
    for (int thread_y = 0; thread_y < THREAD_SIZE_Y; thread_y += 4) {
      FETCH_FLOAT4(frag_a[0][thread_y]) = FETCH_FLOAT4(
          A_s[load_stage_idx ^ 1][0][THREAD_SIZE_Y * ty + thread_y]);
    }
    // from B_s to frag_b
    for (int thread_x = 0; thread_x < THREAD_SIZE_X; thread_x += 4) {
      FETCH_FLOAT4(frag_b[0][thread_x]) = FETCH_FLOAT4(
          B_s[load_stage_idx ^ 1][0][THREAD_SIZE_X * tx + thread_x]);
    }
// compute THREAD_SIZE_X * THREAD_SIZE_Y
#pragma unroll
    for (int thread_y = 0; thread_y < THREAD_SIZE_Y; thread_y++) {
#pragma unroll
      for (int thread_x = 0; thread_x < THREAD_SIZE_X; thread_x++) {
        // accum 在每一次的小循环中保存的只是完整解的一部分
        accum[thread_y][thread_x] += frag_a[1][thread_y] * frag_b[1][thread_x];
      }
    }

  } while (tile_idx < K);
// 完成大循环后，该block中该线程负责的rm * rn的数据已经计算完成
#pragma unroll
  for (int thread_y = 0; thread_y < THREAD_SIZE_Y; thread_y++) {
    for (int thread_x = 0; thread_x < THREAD_SIZE_X; thread_x += 4) {
      FETCH_FLOAT4(
          C[OFFSET(BLOCK_SIZE_M * by + ty * THREAD_SIZE_Y + thread_y,
                   BLOCK_SIZE_N * bx + tx * THREAD_SIZE_X + thread_x, N)]) =
          FETCH_FLOAT4(accum[thread_y][thread_x]);
    }
  }
}