/**
 * @file reduction.cu
 * @author aklice
 * @brief 通过让每个线程在多计算一些元素，来提高线程的利用率
 * @version 0.1
 * @date 2024-03-28
 *
 * @copyright Copyright (c) 2024
 *
 */
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>

#include <cstdlib>

#define THREAD_PER_BLOCK 256
#define CEIL_DIV(x, y) (((x) + (y)-1) / (y))
#define checkCudaErrors(func)                                                  \
  {                                                                            \
    hipError_t err = (func);                                                  \
    if (err != hipSuccess)                                                    \
      printf("%s %d CUDA: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
  }

__global__ void reduction_kernel0(float* A, float* out) {
  int bx = blockIdx.x;
  int tx = threadIdx.x;
  int tid = bx * blockDim.x * 2 + tx;  // 这里 blockDim.x * 2 = NUM_PER_BLOCK
  __shared__ float A_s[THREAD_PER_BLOCK];
  A_s[tx] = A[tid] + A[tid + blockDim.x];
  __syncthreads();
  for (int stride = blockDim.x / 2; stride >= 1; stride /= 2) {
    if (tx < stride) {
      A_s[tx] += A_s[tx + stride];
    }
    __syncthreads();
  }

  if (tx == 0) {
    out[bx] = A_s[tx];
  }
}

int main() {
  const int N = 32 * 1024 * 1024;
  float* h_A;
  h_A = (float*)malloc(N * sizeof(float));
  float* d_A;
  checkCudaErrors(hipMalloc((void**)&d_A, N * sizeof(float)));
  int NUM_PER_BLOCK = 2 * THREAD_PER_BLOCK;
  int block_num = N / NUM_PER_BLOCK;
  float* d_out;
  checkCudaErrors(hipMalloc((void**)&d_out, block_num * sizeof(float)));
  float *h_out, *res;
  h_out = (float*)malloc(block_num * sizeof(float));
  res = (float*)malloc(block_num * sizeof(float));

  for (int i = 0; i < N; i++) {
    h_A[i] = 1;
  }
  for (int i = 0; i < block_num; i++) {
    int sum = 0;
    for (int j = 0; j < NUM_PER_BLOCK; j++) {
      sum += h_A[i * NUM_PER_BLOCK + j];
    }
    res[i] = sum;
  }
  checkCudaErrors(
      hipMemcpy(d_A, h_A, N * sizeof(float), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_out, h_out, block_num * sizeof(float),
                             hipMemcpyHostToDevice));

  dim3 threads_per_block = THREAD_PER_BLOCK;
  dim3 block_per_grid = CEIL_DIV(N, NUM_PER_BLOCK);
  int nIter = 1000;
  hipEvent_t start, stop;
  checkCudaErrors(hipEventCreate(&start));
  checkCudaErrors(hipEventCreate(&stop));
  checkCudaErrors(hipEventRecord(start, 0));
  for (int i = 0; i < nIter; i++) {
    reduction_kernel0<<<block_per_grid, threads_per_block>>>(d_A, d_out);
  }
  checkCudaErrors(hipEventRecord(stop, 0));
  checkCudaErrors(hipEventSynchronize(stop));
  float msec = 0.f;
  checkCudaErrors(hipEventElapsedTime(&msec, start, stop));
  checkCudaErrors(hipMemcpy(h_out, d_out, block_num * sizeof(float),
                             hipMemcpyDeviceToHost));

  float eps = 1.e-6;  // machine zero
  bool flag = true;
  for (int i = 0; i < block_num; i++) {
    if (fabs(h_out[i] - res[i]) > eps) {
      printf("Error in  %d  Answer = %f, Res = %f.\n", i, h_out[i], res[i]);
      flag = false;
      break;
    }
  }
  if (flag) {
    printf("Calculate Successed!\n");
  }
  int bytes = N * sizeof(float);
  float bandwidth = bytes / (msec * 1e6 / nIter);

  printf("Bandwidth is %.3f\n", bandwidth);

  hipFree(d_A);
  hipFree(d_out);
  free(h_A);
  free(h_out);
  free(res);
  return 0;
}